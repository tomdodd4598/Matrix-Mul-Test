#include "gpu_impl.h"
#include "gpu_utils.h"
#include "type_utils.h"

#include <hipblas.h>
#include <hip/hip_complex.h>

#include <complex>
#include <vector>

void gpu_matmul(usize dim, std::vector<complex> const& mat_a, std::vector<complex> const& mat_b, std::vector<complex>& mat_c) {
    hipDoubleComplex* d_mat_a;
    hipDoubleComplex* d_mat_b;
    hipDoubleComplex* d_mat_c;

    const auto bytes = dim * dim * sizeof(complex);

    CHECK_CUDA(hipMalloc(&d_mat_a, bytes));
    CHECK_CUDA(hipMalloc(&d_mat_b, bytes));
    CHECK_CUDA(hipMalloc(&d_mat_c, bytes));

    CHECK_CUDA(hipMemcpy(d_mat_a, mat_a.data(), bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_mat_b, mat_b.data(), bytes, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    const auto alpha = make_hipDoubleComplex(1.0, 0.0);
    const auto beta  = make_hipDoubleComplex(0.0, 0.0);

    CHECK_CUBLAS(hipblasZgemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        dim, dim, dim,
        &alpha,
        d_mat_b, dim,
        d_mat_a, dim,
        &beta,
        d_mat_c, dim
    ));

    CHECK_CUBLAS(hipblasDestroy(handle));

    CHECK_CUDA(hipMemcpy(mat_c.data(), d_mat_c, bytes, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_mat_a));
    CHECK_CUDA(hipFree(d_mat_b));
    CHECK_CUDA(hipFree(d_mat_c));
}

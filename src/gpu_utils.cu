#include "gpu_utils.h"
#include "type_utils.h"

#include <hip/hip_runtime.h>

#include <vector>

const std::vector<int> gpu_ids = get_gpu_ids();
const usize gpu_count = gpu_ids.size();

// Modified from QuEST
std::vector<int> get_gpu_ids() {
    int device_count;
    hipError_t cudaResultCode = hipGetDeviceCount(&device_count);
    if (cudaResultCode != hipSuccess) {
        device_count = 0;
    }

    std::vector<int> gpu_ids;
    hipDeviceProp_t properties;
    for (int i = 0; i < device_count; ++i) {
        hipGetDeviceProperties(&properties, i);
        if (properties.major != 9999) {
            gpu_ids.push_back(i);
        }
    }

    return gpu_ids;
}

int gpu_get_id() {
    int id;
    hipGetDevice(&id);
    return id;
}

usize gpu_get_free_memory() {
    usize free_memory, total_memory;
    hipMemGetInfo(&free_memory, &total_memory);
    return free_memory;
}

usize gpu_get_max_threads() {
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, gpu_get_id());
    return props.multiProcessorCount * props.maxThreadsPerMultiProcessor;
}
